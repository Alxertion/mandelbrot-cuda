#include "hip/hip_runtime.h"
#include "Cuda.hpp"

#if _DEBUG
#   define cudaCall(cuda_func, ...) { hipError_t status = cuda_func(__VA_ARGS__); cudaAssert((status), __FILE__, #cuda_func, __LINE__); }
#else
#   define cudaCall(cuda_func, ...) { hipError_t status = cuda_func(__VA_ARGS__); }
#endif

inline void cudaAssert(hipError_t status, const char *file, const char* func, int line)
{
    if (status != hipSuccess)
    {
        std::stringstream ss;
        ss << "Error: " << hipGetErrorString(status) << std::endl;
        ss << "Func: " << func << std::endl;
        ss << "File: " << file << std::endl;
        ss << "Line: " << line << std::endl;

        throw std::runtime_error(ss.str());
    }
}

__constant__ Image::pixel_t pixel_colour[16] = 
{
    { 66,  30,  15 },{ 25,   7,  26 },{ 9,   1,  47 },{ 4,   4,  73 },
    { 0,   7, 100 },{ 12,  44, 138 },{ 24,  82, 177 },{ 57, 125, 209 },
    { 134, 181, 229 },{ 211, 236, 248 },{ 241, 233, 191 },{ 248, 201,  95 },
    { 255, 170,   0 },{ 204, 128,   0 },{ 153,  87,   0 },{ 106,  52,   3 }
};

__global__ void mandelbrot_kernel(Image::pixel_t* image, const int width, const int height, const double scale, const double cx, const double cy)
{    
    const int i = threadIdx.y + blockIdx.y * blockDim.y;
    const int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= height || j >= width)
    {
        return;
    }

    const std::uint8_t max_iter = 255;
    const double y = (i - (height >> 1)) * scale + cy;
    const double x = (j - (width >> 1)) * scale + cx;

    double zx = hypot(x - 0.25, y);

    if (x < zx - 2.0 * zx * zx + 0.25 || (x + 1.0) * (x + 1.0) + y * y < 0.0625)
    {
        return;
    }

    std::uint8_t iter = 0;
    double zy, zx2, zy2;
    zx = zy = zx2 = zy2 = 0.0;

    do {
        zy = 2.0 * zx * zy + y;
        zx = zx2 - zy2 + x;
        zx2 = zx * zx;
        zy2 = zy * zy;
    } while (iter++ < max_iter && zx2 + zy2 < 4.0);

    if (iter > 0 && iter < max_iter)
    {
        const std::uint8_t colour_idx = iter % 16;

        image[i * width + j] = pixel_colour[colour_idx];
    }
}

namespace cuda
{
    template<class T, typename... A>
    float launch_kernel(T& kernel, dim3 work, A&&... args)
    {
        int device;
        hipDeviceProp_t props;
        hipGetDevice(&device);
        hipGetDeviceProperties(&props, device);

        int threadBlocks;
        if (props.major == 2)
        {
            threadBlocks = 8;
        }
        else if (props.major == 3)
        {
            threadBlocks = 16;
        }
        else
        {
            threadBlocks = 32;
        }

        int blockSize;
        std::uint32_t minGridSize;
        hipOccupancyMaxPotentialBlockSize((int*)&minGridSize, &blockSize, kernel, 0, 0);

        int maxActiveBlocks = 0;
        do
        {
            hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, kernel, blockSize, 0);

            if (blockSize < props.warpSize || maxActiveBlocks >= threadBlocks)
            {
                break;
            }          

            blockSize -= props.warpSize;
        } 
        while (true);

        int blockSizeDimX, blockSizeDimY;
        blockSizeDimX = blockSizeDimY = (int)pow(2, ceil(log(sqrt(blockSize)) / log(2)));

        while (blockSizeDimX * blockSizeDimY > blockSize)
        {
            blockSizeDimY--;
        }

        dim3 block(blockSizeDimX, blockSizeDimY);
        dim3 grid((work.x + block.x - 1) / block.x, (work.y + block.y - 1) / block.y);
        grid.x = grid.x > minGridSize ? grid.x : minGridSize;
        grid.y = grid.y > minGridSize ? grid.y : minGridSize;

#ifdef _DEBUG
        float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / (float)(props.maxThreadsPerMultiProcessor / props.warpSize);

        std::cout << "Grid of size " << grid.x * grid.y << std::endl;
        std::cout << "Launched blocks of size " << blockSize << std::endl;
        std::cout << "Theoretical occupancy " << occupancy * 100.0f << "%" << std::endl;
#endif

        hipEvent_t start;
        cudaCall(hipEventCreate, &start);

        hipEvent_t stop;
        cudaCall(hipEventCreate, &stop);

        cudaCall(hipEventRecord, start, 0);

        kernel<<< grid, block >>>(std::forward<A>(args)...);

        cudaCall(hipGetLastError);
        cudaCall(hipEventRecord, stop, 0);
        cudaCall(hipEventSynchronize, stop);

        float elapsed_time;
        cudaCall(hipEventElapsedTime, &elapsed_time, start, stop);

        cudaCall(hipEventDestroy, start);
        cudaCall(hipEventDestroy, stop);

        hipProfilerStop();

        return elapsed_time;
    }

    std::uint64_t generate_mandelbrot(Image& image, double cx, double cy)
    {
        Image::pixel_t* d_img_data;
        const int img_size = image.width * image.height * sizeof(Image::pixel_t);
        const double scale = 1.0 / (image.width / 4.0);

        cudaCall(hipMalloc, (void**)&d_img_data, img_size);
        cudaCall(hipMemset, d_img_data, 0, img_size);

        float elapsed_time = launch_kernel(mandelbrot_kernel, dim3(image.width, image.height), d_img_data, image.width, image.height, scale, cx, cy);

        cudaCall(hipMemcpy, &image.data[0], d_img_data, img_size, hipMemcpyDeviceToHost);

        cudaCall(hipFree, d_img_data);

        return static_cast<std::uint64_t>(elapsed_time);
    }
}